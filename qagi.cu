#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>

#define NEGATIVE_INF -1 //(-inf, b)
#define POSITIVE_INF 1 //(a, inf)
#define BOTH_INF 2 //(-inf, inf)
#define MAX_ITERATIONS 50 //Number of cycles allowed before quit
#define MAX_SUBINTERVALS_ALLOWED 10
#define MAX_TOTALDIVISIONS_ALLOWED 10

#define ABS(x) ((x < 0) ? -x : x)
#define MAX(x, y) ((x < y) ? y : x)
#define MIN(x, y) ((x < y) ? x : y)

enum {
    NORMAL, MAX_ITERATIONS_ALLOWED = 0x1, ROUNDOFF_ERROR = 0x2, BAD_INTEGRAND_BEHAVIOR = 0x4,
    TOLERANCE_CANNOT_BE_ACHIEVED = 0x8, DIVERGENT = 0x10, INVALID_INPUT = 0x20
};

typedef struct sintegral {
    double a; //left end point of interval
    double b; //right end point of interval
    double error; //error over the interval
    double result; //result over the interval
    double resasc; //approximation of F-I/(B-A) over transformed integrand
    double resabs; //approximation of the integral over the absolute value of the function
    unsigned skimmed : 1; //logical variable denoting whether subinterval is skimmed
} Subintegral;

typedef struct extr {
    double list[52]; //Lower two tables of the epsilon table
    double prevlist[3]; //List of three most recent elements
    int index; //Index of epsilon table
    double error; //Error found in epsilon extrapolation
    double result; //Result from epsilon table
    int calls; //Number of calls to the extrapolation procedure
} Epsilontable;

typedef struct inte {
    char equation[50]; //String for equation to be parsed
    int evaluations; //number of evaluations of the integrand
    double result; //total result calculated
    double abserror; //total error in result calculated
    int ier; //bit for error flagging
    int iroff1, iroff2, iroff3; //flags for the amount of round off error detected through three different types
    int extrap; //logical variable denoting whether the algorithm is attempting extrapolation or not
    int noext; //logical variable denoting whether extrapolation is no longer allowed
} Integrand;

typedef struct res {
    Subintegral original;
    Subintegral* results;
    double totalerror;
    double totalresult;
    int divisions;
    int nskimmed; 
} Result;

typedef struct dev {
    Subintegral* list; //list containing left end point, right end points, result, and error estimate 
    Result* result; //list containing results of the singular intervals
    Integrand* integrand; //structure representing the integrand
    double* totalresult; //total results over the list
    double* totalerror; //total error over the list
    int* index; //index on the device side
} Device;

__device__ double f(double x) {
    return  1 / (1 + (x * x));
}

void flagError(Integrand*, int);
void setvalues(Subintegral*, Integrand*, double, int, int);

/*
    Evaluates initial interval using parallelized Gauss-Kronrod
    Quadrature. It then intitializes errorsum and resultsum on
    both host and device side.

    Parameters:
        device - structure meant to represent reusable device memory
        bound - finite bound for semi-infinite integrals.
                Default is 0.
        inf - constant used to denote which direction the
                integral is infinite 
        errorsum - total error over the entire list
        resultsum - total results over the entire list
*/

__global__ void CUDA_qk15i(double, int, Subintegral*);
__global__ void setTotals(Subintegral*, double*, double*);

void fqk15i(Device device, int bound, int inf, double* resultsum, double* errorsum)
{
    /* Perform Initial Gauss-Kronrod Calculation */
    CUDA_qk15i <<<1, 15>>> (bound, inf, device.list);
    /* Copy result and error to device side total error and results */
    setTotals<<<1,1>>>(device.list, device.totalerror, device.totalresult);
    /* Copy result and error to host side total error and results */
    hipMemcpy(resultsum, &device.list[0].result, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(errorsum, &device.list[0].error, sizeof(double), hipMemcpyDeviceToHost);
    
}

/*
    Evaluates entire list of interval using a "water fall" method.
    This method uses multiple threads to evluate each interval and
    determine the appropriate amount of divisions which depends on
    the percentage of error in each interval. Round off error is
    then checked and flagged accordingly. Intervals that have
    a tolerable amount of error are then "skimmed" also known as
    being taken out of circulation.

    Parameters:
        device - structure meant to represent reusable device memory
        integrand - structure meant to represent variables associated
                    with the integrand as a whole
        bound - finite bound for semi-infinite integrals.
                Default is 0.
        inf - constant used to denote which direction the
                integral is infinite 
        index - current index of list
        abserr_thresh - absolute error threshold
        relerr_thresh - relative error threshold
        errorsum - total error over the entire list
        resultsum - total results over the entire list
*/

__global__ void dqk15i(Subintegral*, Result*, int, int, int, int*, double*, double*);
__global__ void checkRoundOff(Integrand*, Result*, int, int*);
__global__ void skimValues(Subintegral*, Result*, int, int*, double, double, double*);

void wqk15i(Device device, Integrand* integrand, int bound, int inf, int* index, double abserr_thresh, double relerr_thresh, double* errorsum, double* resultsum)
{
    int oindex; //Original index before quadrature

    /* Reset necissary arguments */
    oindex = *index;
    hipMemset(device.index, 0, sizeof(int)); //Resets device index for allocating memory

    /* Perform Dynamic Gauss-Kronrod Quadrature */
    dqk15i <<<oindex+1, 1>>> (device.list, device.result, bound, inf, oindex, device.index, device.totalerror, device.totalresult);
    /* Check round off error */
    checkRoundOff <<<oindex+1, 1>>> (device.integrand, device.result, oindex, device.index);
    /* Skim results */
    hipMemset(device.index, 0, sizeof(int)); //Resets device index for allocating memory
    skimValues <<<oindex+1, 1>>> (device.list, device.result, oindex, device.index, abserr_thresh, relerr_thresh, device.totalresult);

    /* Copy results necissary to the CPU side */
    hipMemcpy(index, device.index, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(integrand, device.integrand, sizeof(Integrand), hipMemcpyDeviceToHost);
    hipMemcpy(resultsum, device.totalresult, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(errorsum, device.totalerror, sizeof(double), hipMemcpyDeviceToHost);
}

/*
    Uses multiple threads to launch Gauss-Kronrod Quadrature over
    every interval. Each interval gets a dynamically allocated 
    amount of divisions with each one getting at least 2. These
    results are then appended to rlist and used checkRoundOff and
    skimValues.
    
    Parameters:
        list - list of device sided intervals to be divided
        rlist - list of results from original intervals that are
                used in other kernels
        bound - finite bound for semi-infinite integrals.
                Default is 0.
        inf - constant used to denote which direction the
                integral is infinite 
        oindex - original index before Gauss-Krondrod Quadrature
        nindex - new index after Gauss-Kronrod Quadrature
        errorsum - total error over the entire list
        resultsum - total results over the entire list
*/

__global__ void qk15i(Subintegral, Subintegral*, int, int, int);
__device__ int findDivisions(double, double, int, int);
__device__ double sumResults(Subintegral*, int);
__device__ double sumError(Subintegral*, int);
__device__ Subintegral* alloclist(Subintegral*, int*, int);
__device__ double dbl_atomicAdd(double*, double);

/* Memory to be dynamically allocated */
__device__ Subintegral allocmem[MAX_SUBINTERVALS_ALLOWED];

__global__ void dqk15i(Subintegral* list, Result* rlist, int bound, int inf, int oindex, int* nindex, double* errorsum, double* resultsum)
{
    int tindex; //Unique Thread index
    int divisions; //Amount of divisions allocated to subinterval
    double toterror; //Total error over subinterval
    double totresult; //Total result over the interval
    Subintegral original; //Original interval before divisions
    Subintegral* memindex; //Position in global memory to return results

    tindex = threadIdx.x + blockIdx.x * blockDim.x;

    if (tindex <= oindex) {
        original = list[tindex];
        /* Find the amount of divisions and allocate amount of corresponding memory */
        divisions = findDivisions(list[tindex].error, *errorsum, oindex, MAX_TOTALDIVISIONS_ALLOWED);
        memindex = alloclist(allocmem, nindex, divisions); 
        /* Perform Dynamic Gauss-Kronrod Quadrature*/
        qk15i <<<divisions, 1>>> (original, memindex, bound, inf, divisions);
        hipDeviceSynchronize();
    
    /* Improve previous approximations to integral and error and test for accuracy  */
        totresult = sumResults(memindex, divisions);
        toterror = sumError(memindex, divisions);
        dbl_atomicAdd(errorsum, toterror - original.error);
        dbl_atomicAdd(resultsum, totresult - original.result);
    /* Append results to interfunctional list */
        rlist[tindex].original = original;
        rlist[tindex].results = memindex;
        rlist[tindex].totalresult = totresult;
        rlist[tindex].totalerror = toterror;
        rlist[tindex].divisions = divisions;
        rlist[tindex].nskimmed = 0;
    }
}

/*
    Uses multiple threads to evaluate round of error in results
    found from each interval divided in dqk15i.

    Parameters:
        integrand - structure meant to represent variables associated
                    with the integrand as a whole
        rlist - list of results from original intervals that are
                used in other kernels
        oindex - original index of list
        nindex - new index of list
*/ 
__device__ int checkRO(Subintegral*, int);

__global__ void checkRoundOff(Integrand* integrand, Result* rlist, int oindex, int* nindex) 
{
    int tindex; //Unique Thread index
    int divisions; //Amount of divisions allocated to subinterval
    double toterror; //Total error over subinterval
    double totresult; //Total result over the interval
    Subintegral original; //Original subintegral split
    Subintegral* list; //Position in global memory to return results

    tindex = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tindex <= oindex) {
        divisions = rlist[tindex].divisions;
        toterror = rlist[tindex].totalerror;
        totresult = rlist[tindex].totalresult;
        list = rlist[tindex].results;
        original = rlist[tindex].original;
        /* Checking roundoff error */
        if (checkRO(list, divisions)) {
            if (ABS(original.result - totresult) <= 1.0E-05 * ABS(totresult)
                && 9.9E-01 * original.error <= toterror)
                if (integrand->extrap)
                    atomicAdd(&(integrand->iroff2), 1);
                else
                    atomicAdd(&(integrand->iroff1), 1);
            if (*nindex > 10 && original.error < toterror)
                atomicAdd(&(integrand->iroff3), 1);
        }
    }
}

/*
    Used to apply multiple Gauss-Kronrod Quadratures over
    one interval. The amount is dependent on amount of
    divisions allocated in dqk15i. First divides initial
    interval, performs Gauss-Kronrod Quadrature, then puts
    result into array of Subintegrals. 

    Parameters:
        intitial - intitial interval to be divided
        list - list to return results to
        bound - finite bound for semi-infinite integrals.
                Default is 0.
        inf - constant used to denote which direction the
                integral is infinite 
        divisions - number of divisions to be done on the
                    initial interval
*/

__global__ void qk15i(Subintegral initial, Subintegral* list, int bound, int inf, int divisions)
{
    double delx; //the distance inbetween each point evaluated
    int tindex; //unique thread identifier

    tindex = threadIdx.x + blockIdx.x * blockDim.x;
    delx = (initial.b - initial.a) / divisions;
    /* Creating interval to be disected */
    list[tindex].a = delx * tindex; 
    list[tindex].b = delx * (tindex + 1);
    /* Multiple Gauss-Kronrod Quadrature */
    CUDA_qk15i <<<1, 15>>> (bound, inf, list + tindex);
}

/*
    CUDA translated 15-point Gauss Quadrature.

    Parameters:
        bound - finite bound for semi-infinite integrals.
                Default is 0.
        inf - constant used to denote which direction the
                integral is infinite 
        interval - interval to be evaluated
*/
__global__ void CUDA_qk15i(double bound, int inf, Subintegral* interval)
{
    double xk[] = { //arguments for Gauss-Kronod quadrature
        0.0, 9.491079123427585E-01,
        8.648644233597691E-01, 7.415311855993944E-01,
        5.860872354676911E-01, 4.058451513773972E-01,
        2.077849550078985E-01, 9.914553711208126E-01
    };
    double wg[] = { //weight for Gauss rule
        4.179591836734694E-01, 1.294849661688697E-01,
        0.0, 2.797053914892767E-01,
        0.0, 3.818300505051189E-01,
        0.0, 0.0
    };
    double wgk[] = { //weight for Gauss-Kronrod rule
        2.094821410847278E-01, 6.309209262997855E-02,
        1.047900103222502E-01, 1.406532597155259E-01,
        1.690047266392679E-01, 1.903505780647854E-01,
        2.044329400752989E-01, 2.293532201052922E-02
    };

    __shared__ double resultg, resultk, resulta; //results for Gauss and Kronrod rules and the absolute value of Kronrod rule
    __shared__ double resultasc; //the integral of the value of the integral subtracted by the mean value of the integral
    double fval; //function evaluated at transformed arguments
    double x, sx; //arguments for the center of the subintervals and right or left subinterval depending on sign
    double transx; //transformed arguments
    double center, hlength; //Center of transformed integral and half length of integral
    int dinf; //Variable that changes the tranformation equation depending on the orientation of the infinite portion
    int tindex; //Index for thread
    double mean_value; //approximation of mean value over tranformed integrand
    int sign; //Determines if shifted argument is shifted right or left, determined by index

    tindex = threadIdx.x + blockIdx.x * blockDim.x;
    sign = (tindex > 7) ? -1 : 1; //If above 7, shift argument left, else shift right
    tindex -= (tindex > 7) ? 7 : 0; //Index's above 7 share same elements with the index 7 behind it
    dinf = MIN(1, inf);
    hlength = (interval->b - interval->a) / 2;
    center = (interval->a + interval->b) / 2;
    if (tindex == 0) {
        resultg = 0;
        resultk = 0;
        resulta = 0;
        resultasc = 0;
    }
    __syncthreads();

    /* Start computing the 15 point Kronrod estimation */

    x = hlength * xk[tindex]; //Shift center of subinterval
    sx = center + sign * x; //Shift either right or left
    transx = bound + dinf * (1 - sx) / sx; //tranform and evaluate using tranformation equation
    fval = f(transx);
    if (inf == BOTH_INF)
        fval += f(-transx);
    fval /= (sx * sx);
    dbl_atomicAdd(&resultg, wg[tindex] * fval);
    dbl_atomicAdd(&resultk, wgk[tindex] * fval);
    dbl_atomicAdd(&resulta, wgk[tindex] * ABS(fval));
    __syncthreads();

    /* Calculate resasc */
    mean_value = resultk / 2;
    dbl_atomicAdd(&resultasc, wgk[tindex] * ABS(fval - mean_value));
    __syncthreads();

    if (tindex == 0) {
        interval->result = resultk * hlength;
        interval->resasc = resultasc * hlength;
        interval->resabs = resulta * hlength;
        interval->skimmed = 0;

        /* Calculating error */
        interval->error = ABS((resultk - resultg) * hlength);

        if (interval->resasc != 0 && interval->error != 0) //traditonal way to calculate error
            interval->error = interval->resasc * MIN(1, pow(200 * interval->error / interval->resasc, 1.5));
        if (interval->resabs > DBL_MIN / (DBL_EPSILON * 50)) //Checks roundoff error
            interval->error = MAX((DBL_EPSILON / 50) * interval->resabs, interval->error);
    }
}
/*
    Finds the appropriate amount of divisions to be allocated
    depending on percentage of error in interval.

    Parameters:
        error - amount of error in interval
        errorsum - total error over entire list
        index - current index of list
        maxallowed - maximum divisions allowed over
                     entire list
*/
__device__ int findDivisions(double error, double errorsum, int index, int maxallowed) {

    int allowed = maxallowed - ((index + 1) * 2); //Amount of extra divisions to be distributed  
    return (int) ((error / errorsum) * allowed) + 2; //Gives out a default of 2 threads and gives excess to intervals with high error

}

/*
    Sums all results calulated.

    Parameters:
        results - list of calculated intervals
        num - amount of intervals
*/
__device__ double sumResults(Subintegral* results, int num)
{
    double res = 0;
    for (int i = 0; i < num; i++)
        res += results[i].result;
    return res;
}

/*
    Sums all error found.

    Parameters:
        results - list of calculated intervals
        num - amount of intervals
*/
__device__ double sumError(Subintegral* results, int num)
{
    double err = 0;
    for (int i = 0; i < num; i++)
        err += results[i].error;
    return err;
}

/*
    Checks if every member in list satisfies this
    condition: resasc == error

    Parameters:
        results - list of calculated intervals
        num - amount of intervals
*/
__device__ int checkRO(Subintegral* results, int num)
{
    for (int i = 0; i < num; i++)
        if (results[i].resasc == results[i].error)
            return 0;
    return 1;
}

/* 
    Allocates space in a list.

    Parameters:
        list - list to be allocated
        index - current index of list
        amount - amount of space needed
*/
__device__ Subintegral* alloclist(Subintegral* list, int* index, int amount)
{
    int old; //Older copy of variable
    int mindex;

    old = *index;
    do {
        mindex = old;
        old = atomicCAS(index, mindex, mindex + amount);
    } while (mindex != old);
    return mindex + list;
}

/*
    Flags list of results to be "skimmed" or also known as
    taking an interval out of circulation.

    Parameters:
        results - results to be flagged
        index - current index of list
        errorbound - tolerable error over total
                     error in list
        nskimmed - number of skimmed results found
*/
__global__ void flag(Subintegral* results, int index, double errorbound, int* nskimmed) 
{
    int tindex; //Unique thread identifier

    tindex = threadIdx.x + blockIdx.x * blockDim.x;

    /* If results within toleration, mark them for skimming */
    if (tindex <= index && results[tindex].error <= errorbound * (results[tindex].b - results[tindex].a)) {
        results[tindex].skimmed = 1;
        atomicAdd(nskimmed, 1);
    }
}

/*
    Skims values from results and returns them to
    the main list.

    Parameters:
        list - main list to returb intervals to
        results - struct representing important variables to each
                  divided interval
        oindex - original index before Gauss-Kronrod Quadrature
        nindex - new index after Gauss-Kronrod Quadrature and skimming
        abserr_thresh - absolute error threshold
        relerr_thresh - relative error threshold
        errorsum - total error over the entire list
        resultsum - total results over the entire list
*/
        
__global__ void skimValues(Subintegral* list, Result* results, int oindex, int* nindex, double abserr_thresh, double relerr_thresh, double* resultsum)
{
    int tindex; //Unique thread identifier
    int slength; //Length of skimmed list
    int length; //Length of the non-skimmed list
    Subintegral* slist; //Skimmed list
    Subintegral* nslist; //Non-Skimmed list
    double errorbound; //Error bound used for flagging intervals

    tindex = threadIdx.x + blockIdx.x * blockDim.x;

    if (tindex <= oindex) {
        length = results[tindex].divisions;
        /* Start flagging results */
        errorbound = MAX(abserr_thresh, relerr_thresh * ABS(*resultsum));
        flag <<<length, 1>>> (results[tindex].results, length, errorbound, &results[tindex].nskimmed);
        nslist = results[tindex].results;
        slength = length - results[tindex].nskimmed;
        if (slength == 0) {
            /* Find Positions in Global Memory */
            slist = alloclist(list, nindex, slength);
            /* Place intervals that aren't skimmed into global */
            while (slength > 0) {
                if (!nslist[length].skimmed) {
                    slist[slength-1] = nslist[length-1];
                    slength--; length--;
                }
            }
        }
    }
}

/*
    Kernel used to set the reuseable device memory of
    totalerror and totalresults to the result and error
    of the first intitial value after fqk15i.

    Parameters:
        list - device sided list to hold subintegrals
        totalerror - device sided memory for total error
                     in the list
        totalresult - device sided memory for total results
                      in the list
*/

__global__ void setTotals(Subintegral* list, double* totalerror, double* totalresult) 
{
    *totalerror = list[0].error;
    *totalresult = list[0].result;
}

/*
    Kernel used to set the first interval to boundary of
    (0,1). Used in fqk15i.

    Parameters:
        list - device sided list to hold subintegrals
*/
__global__ void setInterval(Subintegral* list) 
{
    list[0].a = 0;
    list[0].b = 1;
}

/*
    Function used to preform addition atomically with
    double precision.
    Parameters:
        address - pointer to memory address of double to be incremented
        val - the value to increment with
*/
__device__ double dbl_atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
            __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

/*
    Function turns on bits to flag errors over
    the integrand.
    Parameters:
        integrand - the structure repesenting the integrand
        error - desired error to be flagged
*/
void flagError(Integrand* integrand, int error)
{
    integrand->ier |= error;
}


/*
    Function used to finish up the program by setting the correct
    values to the integrand
    
    Parameters:
        list - list of subintervals bisected
        integrand - structure representing the bundle of variables associated with
                    the integrand
        index - current index of the list
        inf - constant denoting which direction the integral
              is infinite 
*/
void setvalues(Subintegral* list, Integrand* integrand, double errorsum, int index, int inf)
{
    double res = 0;
    for (int i = 0; i <= index; i++)
        res += list[i].result;
    integrand->result = res;
    integrand->evaluations = (inf == BOTH_INF) ? 2 * (15 + index * 30) : 15 + index * 30;
    integrand->abserror = errorsum;
}

int main()
{
    Integrand integrand;
    Device device;
    int index;
    double errorsum;
    double resultsum;

    integrand.ier = 0;
    integrand.evaluations = 0;
    integrand.result = 0;
    integrand.abserror = 0;
    index = 0;

    /* Allocate device side memory */
    Subintegral* d_list; hipMalloc((void**) &d_list, sizeof(Subintegral) * MAX_SUBINTERVALS_ALLOWED);
    Result* d_results; hipMalloc((void**) &d_results, sizeof(Result) * MAX_SUBINTERVALS_ALLOWED);
    Integrand* d_integrand; hipMalloc((void**) &d_integrand, sizeof(Integrand));
    double* d_toterror;hipMalloc((void**) &d_toterror, sizeof(double));
    double* d_totresult; hipMalloc((void**) &d_totresult, sizeof(double));
    int* d_index; hipMalloc((void**) &d_index, sizeof(double));
    device.list = d_list;
    device.result = d_results;
    device.integrand = d_integrand;
    device.totalerror = d_toterror;
    device.totalresult = d_totresult;
    device.index = d_index;
    hipMemcpy(device.integrand, &integrand, sizeof(Integrand), hipMemcpyHostToDevice);
    /* Set first interval to (1,0) */
    setInterval<<<1,1>>>(device.list);
    /* Parallel Gauss-Kronrod Quadrature */
    fqk15i(device, 0, BOTH_INF, &resultsum, &errorsum);

    wqk15i(device, &integrand, 0, BOTH_INF, &index, 0, 0, &errorsum, &resultsum);
    Subintegral list[MAX_SUBINTERVALS_ALLOWED];
    hipMemcpy(list, device.list, sizeof(Subintegral) * MAX_SUBINTERVALS_ALLOWED, hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    printf("%s\n", hipGetErrorString(error));
    printf("%d\n", index);
    for (int i = 0; i < index; i++)
        printf("%f %f\n", list[i].result, list[i].error);
    printf("\n");
    printf("%f %f\n", resultsum, errorsum);
}
